#include "hip/hip_runtime.h"
#include "vgg16_cuda.h"
__global__ void normalize(const uint8_t* const image, float* input, int B, int C, int H, int W){
  float max_int = 255.0L;
  float mean = 0.5L;
  float var = 0.5L;
  int b, c, h, w, i;
  b = blockIdx.x;
  c = blockIdx.y;
  h = threadIdx.y;
  w = threadIdx.x;
  i = b * (C*H*W) + c * (H*W) + h * (W) + w;
  if(h < H && w < W && i < B * C * H * W)
    input[i] = (image[i] / max_int - mean) / var;
}

__global__ void pad(float* input, float* input_padded, int B, int C, int H, int W, int P){
  int H_OUT = H+2*P;
  int W_OUT = W+2*P;
  int b, c, h, w, i, j;
  b = blockIdx.x;
  c = blockIdx.y;
  h = threadIdx.y;
  w = threadIdx.x;
  i = b * (C*H*W) + c * (H*W) + h * (W) + w;
  j = b * (C*H_OUT*W_OUT) + c * (H_OUT*W_OUT) + (h+P) * (W_OUT) + (w+P);
  if(h < H && w < W && i < B * C * H * W)
    input_padded[j] = input[i];
}

__global__ void pad5(float* input, float* input_padded, int B, int C, int H, int W, int P){
  // output shape (4x4)
  // 0 0 0 0
  // 0 1 2 0
  // 0 3 4 0
  // 0 0 0 0
  int H_OUT = H+2*P;
  int W_OUT = W+2*P;
  int b, c;
  b = blockIdx.x;
  c = threadIdx.x;
  int i1 = b * (C*H*W) + c * (H*W) + 0 * (W) + 0;
  int i2 = b * (C*H*W) + c * (H*W) + 0 * (W) + 1;
  int i3 = b * (C*H*W) + c * (H*W) + 1 * (W) + 0;
  int i4 = b * (C*H*W) + c * (H*W) + 1 * (W) + 1;
  int j1 = b * (C*H_OUT*W_OUT) + c * (H_OUT*W_OUT) + (0+P) * (W_OUT) + (0+P);
  int j2 = b * (C*H_OUT*W_OUT) + c * (H_OUT*W_OUT) + (0+P) * (W_OUT) + (1+P);
  int j3 = b * (C*H_OUT*W_OUT) + c * (H_OUT*W_OUT) + (1+P) * (W_OUT) + (0+P);
  int j4 = b * (C*H_OUT*W_OUT) + c * (H_OUT*W_OUT) + (1+P) * (W_OUT) + (1+P);
  input_padded[j1] = input[i1];
  input_padded[j2] = input[i2];
  input_padded[j3] = input[i3];
  input_padded[j4] = input[i4];
}
template <int InputChannel>
__global__ void conv(float* input, float* output, float* weight, float* bias, int B, int H, int W, int IC, int OC, int K){
  int H_OUT = H - (K-1);
  int W_OUT = W - (K-1);
  int b, h, w, oc, j;
  b = blockIdx.x;
  oc = blockIdx.y;
  h = threadIdx.y;
  w = threadIdx.x;
  j = b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + h * W_OUT + w;
  if(h < H_OUT && w < W_OUT && j < B * OC * H_OUT * W_OUT){
    float temp = bias[oc];
    if(InputChannel == 3){
    for(int ic=0; ic<3; ic++){
      int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
      int kernel_base = oc * (IC * K * K) + ic * (K * K);
      for(int kh=0; kh<3; kh++){
	for(int kw=0; kw<3; kw++){
          temp += input[input_base + kh * (W) + kw] * weight[kernel_base + kh * (K) + kw];
	}
      }
    }
    }
    if(InputChannel == 64){
    for(int ic=0; ic<64; ic++){
      int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
      int kernel_base = oc * (IC * K * K) + ic * (K * K);
      for(int kh=0; kh<3; kh++){
	for(int kw=0; kw<3; kw++){
          temp += input[input_base + kh * (W) + kw] * weight[kernel_base + kh * (K) + kw];
	}
      }
    }
    }
    if(InputChannel == 128){
    for(int ic=0; ic<128; ic++){
      int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
      int kernel_base = oc * (IC * K * K) + ic * (K * K);
      for(int kh=0; kh<3; kh++){
	for(int kw=0; kw<3; kw++){
          temp += input[input_base + kh * (W) + kw] * weight[kernel_base + kh * (K) + kw];
	}
      }
    }
    }
    if(InputChannel == 256){
    for(int ic=0; ic<256; ic++){
      int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
      int kernel_base = oc * (IC * K * K) + ic * (K * K);
      for(int kh=0; kh<3; kh++){
	for(int kw=0; kw<3; kw++){
          temp += input[input_base + kh * (W) + kw] * weight[kernel_base + kh * (K) + kw];
	}
      }
    }
    }
    if(InputChannel == 512){
    for(int ic=0; ic<512; ic++){
      int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
      int kernel_base = oc * (IC * K * K) + ic * (K * K);
      for(int kh=0; kh<3; kh++){
	for(int kw=0; kw<3; kw++){
          temp += input[input_base + kh * (W) + kw] * weight[kernel_base + kh * (K) + kw];
	}
      }
    }
    }
    output[j] = (temp > (float)0.0 ? temp : (float)0.0); // relu
  }
}

__global__ void conv5(float* input, float* output, float* weight, float* bias, int B, int H, int W, int IC, int OC, int K){
  // output shape(2x2)
  // 1 2
  // 3 4
  int H_OUT = H - (K-1);
  int W_OUT = W - (K-1);
  int b, oc;
  b = blockIdx.x;
  oc = threadIdx.x;
  float temp1 = bias[oc];
  float temp2 = bias[oc];
  float temp3 = bias[oc];
  float temp4 = bias[oc];
  int ob1 = b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + 0 * W_OUT + 0;
  int ob2 = b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + 0 * W_OUT + 1;
  int ob3 = b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + 1 * W_OUT + 0;
  int ob4 = b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + 1 * W_OUT + 1;
  for(int ic=0; ic<512; ic++){
    int ib1 = b * (IC * H * W) + ic * (H * W) + 0 * (W) + 0;
    int ib2 = b * (IC * H * W) + ic * (H * W) + 0 * (W) + 1;
    int ib3 = b * (IC * H * W) + ic * (H * W) + 1 * (W) + 0;
    int ib4 = b * (IC * H * W) + ic * (H * W) + 1 * (W) + 1;
    int kb = oc * (IC * K * K) + ic * (K * K);
    for(int kh=0; kh<3; kh++){
      for(int kw=0; kw<3; kw++){
	temp1 += input[ib1 + kh * (W) + kw] * weight[kb + kh * (K) + kw];
	temp2 += input[ib2 + kh * (W) + kw] * weight[kb + kh * (K) + kw];
	temp3 += input[ib3 + kh * (W) + kw] * weight[kb + kh * (K) + kw];
	temp4 += input[ib4 + kh * (W) + kw] * weight[kb + kh * (K) + kw];
      }
    }
  }
  output[ob1] = (temp1 > (float)0.0 ? temp1 : (float)0.0); // relu
  output[ob2] = (temp2 > (float)0.0 ? temp2 : (float)0.0); // relu
  output[ob3] = (temp3 > (float)0.0 ? temp3 : (float)0.0); // relu
  output[ob4] = (temp4 > (float)0.0 ? temp4 : (float)0.0); // relu
}

__global__ void pool(float* input, float* output, int B, int C, int H, int W){
  int scale = 2;
  int H_OUT = H / scale;
  int W_OUT = W / scale;
  int b, c, h, w, i, j;
  b = blockIdx.x;
  c = blockIdx.y;
  h = threadIdx.y;
  w = threadIdx.x;
  // i = b * (C*H*W) + c * (H*W) + h * (W) + w;
  j = b * (C*H_OUT*W_OUT) + c * (H_OUT*W_OUT) + h * W_OUT + w; //output base index
  if(h < H_OUT && w < W_OUT && j < B * C * H_OUT * W_OUT){
    float max_val = -255.0;
    for(int sh=0; sh<2; sh++){
      for(int sw=0; sw<2; sw++){
	i = b * (C*H*W) + c * (H*W) + (2*h+sh) * (W) + (2*w+sw); // input base index
        float val = input[i];
	if(val - max_val > (float)0.0)
	  max_val = val;
      }
    }
    output[j] = max_val;
  }
}

__global__ void pool5(float* input, float* output, int B, int C, int H, int W){
  // output shape(1x1)
  // 1
  int scale = 2;
  int H_OUT = H / scale;
  int W_OUT = W / scale;
  int b, c, i, j;
  b = blockIdx.x;
  c = threadIdx.x;
  j = b * (C * H_OUT * W_OUT) + c * (H_OUT * W_OUT);
  float max_val = -255.0;
  for(int sh=0; sh<2; sh++){
    for(int sw=0; sw<2; sw++){
      i = b * (C*H*W) + c * (H*W) + (sh) * (W) + (sw);
      float val = input[i];
      if(val - max_val > (float)0.0)
	max_val = val;
    }
  }
  output[j] = max_val;
}

__global__ void fc(float* input, float* output, float* weight, float* bias, int B, int IC, int OC){
  // 512 --> 10
  int b, oc;
  b = blockIdx.x;
  oc = threadIdx.x;
  if(b * OC + oc < B * OC){
    float temp = bias[oc];
    for(int ic=0; ic<512; ic++)
      temp += weight[oc * IC + ic] * input[b * IC + ic];
    output[b * OC + oc] = temp;
  }
}


void vgg16_cuda::predict(int batch) {
    // Grid: [# of batch, # of output channel, # of tile]
    // Block:[x_size, y_size]
    // ReLU is included in convolution layer
    //#define TILE_WIDTH 8
    
    dim3 dimGrid(batch, input_channel); // 128, 3
    dim3 dimBlock(32, 32);
    normalize<<<dimGrid, dimBlock>>>(d_image, d_input, batch, input_channel, input_size, input_size);
  
    //////////BLOCK 1/////////////////////////////////
    // TODO: Implement pad
    hipMemset(d_input_padded, 0, sizeof(d_input_padded));
    pad<<<dimGrid, dimBlock>>>(d_input, d_input_padded, batch, input_channel, input_size, input_size, conv1_1_padding_size);

    // TODO: Implement conv1_1
    //int tile1 = (input_size / TILE_WIDTH) * (input_size / TILE_WIDTH);
    dim3 dimGrid11(batch, conv1_1_out_channel); // 128, 64
    dim3 dimBlock11(input_size, input_size); // 32, 32
    conv<3><<<dimGrid11, dimBlock11>>>(d_input_padded, d_C1_1_feature_map, d_conv1_1_weight, d_conv1_1_bias, batch, input_size+2*conv1_1_padding_size, input_size+2*conv1_1_padding_size, conv1_1_in_channel, conv1_1_out_channel, conv1_1_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pad
    hipMemset(d_C1_1_feature_map_padded, 0, sizeof(d_C1_1_feature_map_padded));
    pad<<<dimGrid11, dimBlock11>>>(d_C1_1_feature_map, d_C1_1_feature_map_padded, batch, C1_1_channel, C1_1_size, C1_1_size, conv1_2_padding_size);

    // TODO: Implement conv1_2
    dim3 dimGrid12(batch, conv1_2_out_channel); // 128, 64
    dim3 dimBlock12(C1_1_size, C1_1_size); // 32, 32
    conv<64><<<dimGrid12, dimBlock12>>>(d_C1_1_feature_map_padded, d_C1_2_feature_map, d_conv1_2_weight, d_conv1_2_bias, batch, C1_1_size+2*conv1_2_padding_size, C1_1_size+2*conv1_2_padding_size, conv1_2_in_channel, conv1_2_out_channel, conv1_2_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pool
    dim3 dimGrid1p(batch, C1_2_channel); // 128, 64
    dim3 dimBlock1p(16, 16);
    pool<<<dimGrid1p, dimBlock1p>>>(d_C1_2_feature_map, d_S1_feature_map, batch, C1_2_channel, C1_2_size, C1_2_size);
  
    //////////BLOCK 2/////////////////////////////////
    // TODO: Implement pad
    hipMemset(d_S1_feature_map_padded, 0, sizeof(d_S1_feature_map_padded));	    
    pad<<<dimGrid1p, dimBlock1p>>>(d_S1_feature_map, d_S1_feature_map_padded, batch, S1_channel, S1_size, S1_size, conv2_1_padding_size);

    // TODO: Implement conv2_1
    //int tile2 = (S1_size / TILE_WIDTH) * (S1_size / TILE_WIDTH);
    dim3 dimGrid21(batch, conv2_1_out_channel); // 128, 128
    dim3 dimBlock21(S1_size, S1_size); // 16, 16
    conv<64><<<dimGrid21, dimBlock21>>>(d_S1_feature_map_padded, d_C2_1_feature_map, d_conv2_1_weight, d_conv2_1_bias, batch, S1_size+2*conv2_1_padding_size, S1_size+2*conv2_1_padding_size, conv2_1_in_channel, conv2_1_out_channel, conv2_1_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pad
    hipMemset(d_C2_1_feature_map_padded, 0, sizeof(d_C2_1_feature_map_padded));
    pad<<<dimGrid21, dimBlock21>>>(d_C2_1_feature_map, d_C2_1_feature_map_padded, batch, C2_1_channel, C2_1_size, C2_1_size, conv2_2_padding_size);

    // TODO: Implement conv2_2
    dim3 dimGrid22(batch, conv2_2_out_channel); // 128, 128
    dim3 dimBlock22(C2_1_size, C2_1_size); // 16, 16
    conv<128><<<dimGrid22, dimBlock22>>>(d_C2_1_feature_map_padded, d_C2_2_feature_map, d_conv2_2_weight, d_conv2_2_bias, batch, C2_1_size+2*conv2_2_padding_size, C2_1_size+2*conv2_2_padding_size, conv2_2_in_channel, conv2_2_out_channel, conv2_2_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pool
    dim3 dimGrid2p(batch, C2_2_channel); // 128, 128
    dim3 dimBlock2p(8, 8);
    pool<<<dimGrid2p, dimBlock2p>>>(d_C2_2_feature_map, d_S2_feature_map, batch, C2_2_channel, C2_2_size, C2_2_size);
   
    //////////BLOCK 3/////////////////////////////////
    // TODO: Implement pad
    hipMemset(d_S2_feature_map_padded, 0, sizeof(d_S2_feature_map_padded));
    pad<<<dimGrid2p, dimBlock2p>>>(d_S2_feature_map, d_S2_feature_map_padded, batch, S2_channel, S2_size, S2_size, conv3_1_padding_size);

    // TODO: Implement conv3_1
    dim3 dimGrid31(batch, conv3_1_out_channel); // 128, 256
    dim3 dimBlock31(8, 8);
    conv<128><<<dimGrid31, dimBlock31>>>(d_S2_feature_map_padded, d_C3_1_feature_map, d_conv3_1_weight, d_conv3_1_bias, batch, S2_size+2*conv3_1_padding_size, S2_size+2*conv3_1_padding_size, conv3_1_in_channel, conv3_1_out_channel, conv3_1_kernel_size);
    
    // TODO: Implement relu
    // TODO: Implement pad
    hipMemset(d_C3_1_feature_map_padded, 0, sizeof(d_C3_1_feature_map_padded));
    pad<<<dimGrid31, dimBlock31>>>(d_C3_1_feature_map, d_C3_1_feature_map_padded, batch, C3_1_channel, C3_1_size, C3_1_size, conv3_2_padding_size);

    // TODO: Implement conv3_2
    dim3 dimGrid32(batch, conv3_2_out_channel); // 128, 256
    dim3 dimBlock32(8, 8);
    conv<256><<<dimGrid32, dimBlock32>>>(d_C3_1_feature_map_padded, d_C3_2_feature_map, d_conv3_2_weight, d_conv3_2_bias, batch, C3_1_size+2*conv3_2_padding_size, C3_1_size+2*conv3_2_padding_size, conv3_2_in_channel, conv3_2_out_channel, conv3_2_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pad
    hipMemset(d_C3_2_feature_map_padded, 0, sizeof(d_C3_2_feature_map_padded));
    pad<<<dimGrid32, dimBlock32>>>(d_C3_2_feature_map, d_C3_2_feature_map_padded, batch, C3_2_channel, C3_2_size, C3_2_size, conv3_3_padding_size);

    // TODO: Implement conv3_3
    dim3 dimGrid33(batch, conv3_3_out_channel); // 128, 256
    dim3 dimBlock33(8, 8);
    conv<256><<<dimGrid33, dimBlock33>>>(d_C3_2_feature_map_padded, d_C3_3_feature_map, d_conv3_3_weight, d_conv3_3_bias, batch, C3_2_size+2*conv3_3_padding_size, C3_2_size+2*conv3_3_padding_size, conv3_3_in_channel, conv3_3_out_channel, conv3_3_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pool
    dim3 dimGrid3p(batch, C3_3_channel); // 128, 256
    dim3 dimBlock3p(4, 4);
    pool<<<dimGrid3p, dimBlock3p>>>(d_C3_3_feature_map, d_S3_feature_map, batch, C3_3_channel, C3_3_size, C3_3_size);
   
    //////////BLOCK 4/////////////////////////////////
    // TODO: Implement pad
    hipMemset(d_S3_feature_map_padded, 0, sizeof(d_S3_feature_map_padded));
    pad<<<dimGrid3p, dimBlock3p>>>(d_S3_feature_map, d_S3_feature_map_padded, batch, S3_channel, S3_size, S3_size, conv4_1_padding_size);
    
    // TODO: Implement conv4_1
    dim3 dimGrid41(batch, conv4_1_out_channel); // 128, 512
    dim3 dimBlock41(4, 4);
    conv<256><<<dimGrid41, dimBlock41>>>(d_S3_feature_map_padded, d_C4_1_feature_map, d_conv4_1_weight, d_conv4_1_bias, batch, S3_size+2*conv4_1_padding_size, S3_size+2*conv4_1_padding_size, conv4_1_in_channel, conv4_1_out_channel, conv4_1_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pad
    hipMemset(d_C4_1_feature_map_padded, 0, sizeof(d_C4_1_feature_map_padded));
    pad<<<dimGrid41, dimBlock41>>>(d_C4_1_feature_map, d_C4_1_feature_map_padded, batch, C4_1_channel, C4_1_size, C4_1_size, conv4_2_padding_size);

    // TODO: Implement conv4_2
    dim3 dimGrid42(batch, conv4_2_out_channel); // 128, 512
    dim3 dimBlock42(4, 4);
    conv<512><<<dimGrid42, dimBlock42>>>(d_C4_1_feature_map_padded, d_C4_2_feature_map, d_conv4_2_weight, d_conv4_2_bias, batch, C4_1_size+2*conv4_2_padding_size, C4_1_size+2*conv4_2_padding_size, conv4_2_in_channel, conv4_2_out_channel, conv4_2_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pad
    hipMemset(d_C4_2_feature_map_padded, 0, sizeof(d_C4_2_feature_map_padded));
    pad<<<dimGrid42, dimBlock42>>>(d_C4_2_feature_map, d_C4_2_feature_map_padded, batch, C4_2_channel, C4_2_size, C4_2_size, conv4_3_padding_size);

    // TODO: Implement conv4_3
    dim3 dimGrid43(batch, conv4_3_out_channel); // 128, 512
    dim3 dimBlock43(4, 4);
    conv<512><<<dimGrid43, dimBlock43>>>(d_C4_2_feature_map_padded, d_C4_3_feature_map, d_conv4_3_weight, d_conv4_3_bias, batch, C4_2_size+2*conv4_3_padding_size, C4_2_size+2*conv4_3_padding_size, conv4_3_in_channel, conv4_3_out_channel, conv4_3_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pool
    dim3 dimGrid4p(batch, C4_3_channel); // 128, 512
    dim3 dimBlock4p(2, 2);
    pool<<<dimGrid4p, dimBlock4p>>>(d_C4_3_feature_map, d_S4_feature_map, batch, C4_3_channel, C4_3_size, C4_3_size);
   
    //////////BLOCK 5/////////////////////////////////
    // TODO: Implement pad
    hipMemset(d_S4_feature_map_padded, 0, sizeof(d_S4_feature_map_padded));
    pad5<<<batch, C4_3_channel>>>(d_S4_feature_map, d_S4_feature_map_padded, batch, S4_channel, S4_size, S4_size, conv5_1_padding_size);

    // TODO: Implement conv5_1 [128x512x4x4] --> [128x512x2x2]
    conv5<<<batch, conv5_1_out_channel>>>(d_S4_feature_map_padded, d_C5_1_feature_map, d_conv5_1_weight, d_conv5_1_bias, batch, S4_size+2*conv5_1_padding_size, S4_size+2*conv5_1_padding_size, conv5_1_in_channel, conv5_1_out_channel, conv5_1_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pad
    hipMemset(d_C5_1_feature_map_padded, 0, sizeof(d_C5_1_feature_map_padded));
    pad5<<<batch, conv5_1_out_channel>>>(d_C5_1_feature_map, d_C5_1_feature_map_padded, batch, C5_1_channel, C5_1_size, C5_1_size, conv5_2_padding_size);

    // TODO: Implement conv5_2 [128x512x4x4] --> [128x512x2x2]
    conv5<<<batch, conv5_2_out_channel>>>(d_C5_1_feature_map_padded, d_C5_2_feature_map, d_conv5_2_weight, d_conv5_2_bias, batch, C5_1_size+2*conv5_2_padding_size, C5_1_size+2*conv5_2_padding_size, conv5_2_in_channel, conv5_2_out_channel, conv5_2_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pad
    hipMemset(d_C5_2_feature_map_padded, 0, sizeof(d_C5_2_feature_map_padded));
    pad5<<<batch, conv5_2_out_channel>>>(d_C5_2_feature_map, d_C5_2_feature_map_padded, batch, C5_2_channel, C5_2_size, C5_2_size, conv5_3_padding_size);

    // TODO: Implement conv5_3
    conv5<<<batch, conv5_3_out_channel>>>(d_C5_2_feature_map_padded, d_C5_3_feature_map, d_conv5_3_weight, d_conv5_3_bias, batch, C5_2_size+2*conv5_3_padding_size, C5_2_size+2*conv5_3_padding_size, conv5_3_in_channel, conv5_3_out_channel, conv5_3_kernel_size);

    // TODO: Implement relu
    // TODO: Implement pool // [128x512x2x2] --> [128x512x1x1]
    pool5<<<batch, C5_3_channel>>>(d_C5_3_feature_map, d_S5_feature_map, batch, C5_3_channel, C5_3_size, C5_3_size);
  
    // TODO: Implement fc1 [128x512] --> [128x10]
    fc<<<batch, fc1_out_channel>>>(d_S5_feature_map, d_output, d_fc1_weight, d_fc1_bias, batch, fc1_in_channel, fc1_out_channel);
    // TODO: Implement relu --> This step does not exist in cpu version, just skip this.

    /* NOTE: unless you want to make a major change to this class structure, 
    *  you need to write your output to the device memory d_output 
    *  so that classify() can handle the rest.
    */
}

void vgg16_cuda::prepare_device_memory(uint8_t* image) {
  // Alloc Model Parameters

  //////////BLOCK 1/////////////////////////////////
  hipMalloc((void**)&d_conv1_1_weight,
             sizeof(float) * conv1_1_in_channel * conv1_1_out_channel *
                 conv1_1_kernel_size * conv1_1_kernel_size);
  hipMalloc((void**)&d_conv1_1_bias, sizeof(float) * conv1_1_out_channel);
  hipMalloc((void**)&d_conv1_2_weight,
             sizeof(float) * conv1_2_in_channel * conv1_2_out_channel *
                 conv1_2_kernel_size * conv1_2_kernel_size);
  hipMalloc((void**)&d_conv1_2_bias, sizeof(float) * conv1_2_out_channel);

  //////////BLOCK 2/////////////////////////////////
  hipMalloc((void**)&d_conv2_1_weight,
             sizeof(float) * conv2_1_in_channel * conv2_1_out_channel *
                 conv2_1_kernel_size * conv2_1_kernel_size);
  hipMalloc((void**)&d_conv2_1_bias, sizeof(float) * conv2_1_out_channel);
  hipMalloc((void**)&d_conv2_2_weight,
             sizeof(float) * conv2_2_in_channel * conv2_2_out_channel *
                 conv2_2_kernel_size * conv2_2_kernel_size);
  hipMalloc((void**)&d_conv2_2_bias, sizeof(float) * conv2_2_out_channel);

  //////////BLOCK 3/////////////////////////////////
  hipMalloc((void**)&d_conv3_1_weight,
             sizeof(float) * conv3_1_in_channel * conv3_1_out_channel *
                 conv3_1_kernel_size * conv3_1_kernel_size);
  hipMalloc((void**)&d_conv3_1_bias, sizeof(float) * conv3_1_out_channel);
  hipMalloc((void**)&d_conv3_2_weight,
             sizeof(float) * conv3_2_in_channel * conv3_2_out_channel *
                 conv3_2_kernel_size * conv3_2_kernel_size);
  hipMalloc((void**)&d_conv3_2_bias, sizeof(float) * conv3_2_out_channel);
  hipMalloc((void**)&d_conv3_3_weight,
             sizeof(float) * conv3_3_in_channel * conv3_3_out_channel *
                 conv3_3_kernel_size * conv3_3_kernel_size);
  hipMalloc((void**)&d_conv3_3_bias, sizeof(float) * conv3_3_out_channel);

  //////////BLOCK 4/////////////////////////////////
  hipMalloc((void**)&d_conv4_1_weight,
             sizeof(float) * conv4_1_in_channel * conv4_1_out_channel *
                 conv4_1_kernel_size * conv4_1_kernel_size);
  hipMalloc((void**)&d_conv4_1_bias, sizeof(float) * conv4_1_out_channel);
  hipMalloc((void**)&d_conv4_2_weight,
             sizeof(float) * conv4_2_in_channel * conv4_2_out_channel *
                 conv4_2_kernel_size * conv4_2_kernel_size);
  hipMalloc((void**)&d_conv4_2_bias, sizeof(float) * conv4_2_out_channel);
  hipMalloc((void**)&d_conv4_3_weight,
             sizeof(float) * conv4_3_in_channel * conv4_3_out_channel *
                 conv4_3_kernel_size * conv4_3_kernel_size);
  hipMalloc((void**)&d_conv4_3_bias, sizeof(float) * conv4_3_out_channel);

  //////////BLOCK 5/////////////////////////////////
  hipMalloc((void**)&d_conv5_1_weight,
             sizeof(float) * conv5_1_in_channel * conv5_1_out_channel *
                 conv5_1_kernel_size * conv5_1_kernel_size);
  hipMalloc((void**)&d_conv5_1_bias, sizeof(float) * conv5_1_out_channel);
  hipMalloc((void**)&d_conv5_2_weight,
             sizeof(float) * conv5_2_in_channel * conv5_2_out_channel *
                 conv5_2_kernel_size * conv5_2_kernel_size);
  hipMalloc((void**)&d_conv5_2_bias, sizeof(float) * conv5_2_out_channel);
  hipMalloc((void**)&d_conv5_3_weight,
             sizeof(float) * conv5_3_in_channel * conv5_3_out_channel *
                 conv5_3_kernel_size * conv5_3_kernel_size);
  hipMalloc((void**)&d_conv5_3_bias, sizeof(float) * conv5_3_out_channel);

  //////////FC 1////////////////////////////////////
  hipMalloc((void**)&d_fc1_weight,
             sizeof(float) * fc1_in_channel * fc1_out_channel);
  hipMalloc((void**)&d_fc1_bias, sizeof(float) * fc1_out_channel);

  // Alloc Activations
  hipMalloc((void**)&d_image,
             sizeof(uint8_t) * batch * input_size * input_size * input_channel);
  hipMalloc((void**)&d_input,
             sizeof(float) * batch * input_channel * input_size * input_size);

  //////////BLOCK 1/////////////////////////////////
  hipMalloc((void**)&d_input_padded,
             sizeof(float) * batch * input_channel * (input_size+2*conv1_1_padding_size) * (input_size+2*conv1_1_padding_size));
  hipMalloc((void**)&d_C1_1_feature_map,
             sizeof(float) * batch * C1_1_channel * C1_1_size * C1_1_size);
  hipMalloc((void**)&d_C1_1_feature_map_padded,
             sizeof(float) * batch * C1_1_channel * (C1_1_size+2*conv1_2_padding_size) * (C1_1_size+2*conv1_2_padding_size));
  hipMalloc((void**)&d_C1_2_feature_map,
             sizeof(float) * batch * C1_2_channel * C1_2_size * C1_2_size);
  hipMalloc((void**)&d_S1_feature_map,
             sizeof(float) * batch * S1_channel * S1_size * S1_size);

  //////////BLOCK 2/////////////////////////////////
  hipMalloc((void**)&d_S1_feature_map_padded,
             sizeof(float) * batch * S1_channel * (S1_size+2*conv2_1_padding_size) * (S1_size+2*conv2_1_padding_size));
  hipMalloc((void**)&d_C2_1_feature_map,
             sizeof(float) * batch * C2_1_channel * C2_1_size * C2_1_size);
  hipMalloc((void**)&d_C2_1_feature_map_padded,
             sizeof(float) * batch * C2_1_channel * (C2_1_size+2*conv2_2_padding_size) * (C2_1_size+2*conv2_2_padding_size));
  hipMalloc((void**)&d_C2_2_feature_map,
             sizeof(float) * batch * C2_2_channel * C2_2_size * C2_2_size);
  hipMalloc((void**)&d_S2_feature_map,
             sizeof(float) * batch * S2_channel * S2_size * S2_size);

  //////////BLOCK 3/////////////////////////////////
  hipMalloc((void**)&d_S2_feature_map_padded,
             sizeof(float) * batch * S2_channel * (S2_size+2*conv3_1_padding_size) * (S2_size+2*conv3_1_padding_size));
  hipMalloc((void**)&d_C3_1_feature_map,
             sizeof(float) * batch * C3_1_channel * C3_1_size * C3_1_size);
  hipMalloc((void**)&d_C3_1_feature_map_padded,
             sizeof(float) * batch * C3_1_channel * (C3_1_size+2*conv3_2_padding_size) * (C3_1_size+2*conv3_2_padding_size));
  hipMalloc((void**)&d_C3_2_feature_map,
             sizeof(float) * batch * C3_2_channel * C3_2_size * C3_2_size);
  hipMalloc((void**)&d_C3_2_feature_map_padded,
             sizeof(float) * batch * C3_2_channel * (C3_2_size+2*conv3_3_padding_size) * (C3_2_size+2*conv3_3_padding_size));
  hipMalloc((void**)&d_C3_3_feature_map,
             sizeof(float) * batch * C3_3_channel * C3_3_size * C3_3_size);
  hipMalloc((void**)&d_S3_feature_map,
             sizeof(float) * batch * S3_channel * S3_size * S3_size);

  //////////BLOCK 4/////////////////////////////////
  hipMalloc((void**)&d_S3_feature_map_padded,
             sizeof(float) * batch * S3_channel * (S3_size+2*conv4_1_padding_size) * (S3_size+2*conv4_1_padding_size));
  hipMalloc((void**)&d_C4_1_feature_map,
             sizeof(float) * batch * C4_1_channel * C4_1_size * C4_1_size);
  hipMalloc((void**)&d_C4_1_feature_map_padded,
             sizeof(float) * batch * C4_1_channel * (C4_1_size+2*conv4_2_padding_size) * (C4_1_size+2*conv4_2_padding_size));
  hipMalloc((void**)&d_C4_2_feature_map,
             sizeof(float) * batch * C4_2_channel * C4_2_size * C4_2_size);
  hipMalloc((void**)&d_C4_2_feature_map_padded,
             sizeof(float) * batch * C4_2_channel * (C4_2_size+2*conv4_3_padding_size) * (C4_2_size+2*conv4_3_padding_size));
  hipMalloc((void**)&d_C4_3_feature_map,
             sizeof(float) * batch * C4_3_channel * C4_3_size * C4_3_size);
  hipMalloc((void**)&d_S4_feature_map,
             sizeof(float) * batch * S4_channel * S4_size * S4_size);

  //////////BLOCK 5/////////////////////////////////
  hipMalloc((void**)&d_S4_feature_map_padded,
             sizeof(float) * batch * S4_channel * (S4_size+2*conv5_1_padding_size) * (S4_size+2*conv5_1_padding_size));
  hipMalloc((void**)&d_C5_1_feature_map,
             sizeof(float) * batch * C5_1_channel * C5_1_size * C5_1_size);
  hipMalloc((void**)&d_C5_1_feature_map_padded,
             sizeof(float) * batch * C5_1_channel * (C5_1_size+2*conv5_2_padding_size) * (C5_1_size+2*conv5_2_padding_size));
  hipMalloc((void**)&d_C5_2_feature_map,
             sizeof(float) * batch * C5_2_channel * C5_2_size * C5_2_size);
  hipMalloc((void**)&d_C5_2_feature_map_padded,
             sizeof(float) * batch * C5_2_channel * (C5_2_size+2*conv5_3_padding_size) * (C5_2_size+2*conv5_3_padding_size));
  hipMalloc((void**)&d_C5_3_feature_map,
             sizeof(float) * batch * C5_3_channel * C5_3_size * C5_3_size);
  hipMalloc((void**)&d_S5_feature_map,
             sizeof(float) * batch * S5_channel * S5_size * S5_size);


  hipMalloc((void**)&d_output, sizeof(float) * batch * output_size);

  // Copy Parameters
  //////////BLOCK 1/////////////////////////////////
  hipMemcpy(d_conv1_1_weight, conv1_1_weight,
             sizeof(float) * conv1_1_in_channel * conv1_1_out_channel *
                 conv1_1_kernel_size * conv1_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv1_1_bias, conv1_1_bias, sizeof(float) * conv1_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv1_2_weight, conv1_2_weight,
              sizeof(float) * conv1_2_in_channel * conv1_2_out_channel *
                  conv1_2_kernel_size * conv1_2_kernel_size,
              hipMemcpyHostToDevice);
   hipMemcpy(d_conv1_2_bias, conv1_2_bias, sizeof(float) * conv1_2_out_channel,
              hipMemcpyHostToDevice);

  //////////BLOCK 2/////////////////////////////////
  hipMemcpy(d_conv2_1_weight, conv2_1_weight,
             sizeof(float) * conv2_1_in_channel * conv2_1_out_channel *
                 conv2_1_kernel_size * conv2_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_1_bias, conv2_1_bias, sizeof(float) * conv2_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_2_weight, conv2_2_weight,
              sizeof(float) * conv2_2_in_channel * conv2_2_out_channel *
                  conv2_2_kernel_size * conv2_2_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_2_bias, conv2_2_bias, sizeof(float) * conv2_2_out_channel,
              hipMemcpyHostToDevice);

  //////////BLOCK 3/////////////////////////////////
  hipMemcpy(d_conv3_1_weight, conv3_1_weight,
             sizeof(float) * conv3_1_in_channel * conv3_1_out_channel *
                 conv3_1_kernel_size * conv3_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_1_bias, conv3_1_bias, sizeof(float) * conv3_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_2_weight, conv3_2_weight,
              sizeof(float) * conv3_2_in_channel * conv3_2_out_channel *
                  conv3_2_kernel_size * conv3_2_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_2_bias, conv3_2_bias, sizeof(float) * conv3_2_out_channel,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_3_weight, conv3_3_weight,
              sizeof(float) * conv3_3_in_channel * conv3_3_out_channel *
                  conv3_3_kernel_size * conv3_3_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_3_bias, conv3_3_bias, sizeof(float) * conv3_3_out_channel,
              hipMemcpyHostToDevice);

  //////////BLOCK 4/////////////////////////////////
  hipMemcpy(d_conv4_1_weight, conv4_1_weight,
             sizeof(float) * conv4_1_in_channel * conv4_1_out_channel *
                 conv4_1_kernel_size * conv4_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_1_bias, conv4_1_bias, sizeof(float) * conv4_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_2_weight, conv4_2_weight,
              sizeof(float) * conv4_2_in_channel * conv4_2_out_channel *
                  conv4_2_kernel_size * conv4_2_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_2_bias, conv4_2_bias, sizeof(float) * conv4_2_out_channel,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_3_weight, conv4_3_weight,
              sizeof(float) * conv4_3_in_channel * conv4_3_out_channel *
                  conv4_3_kernel_size * conv4_3_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_3_bias, conv4_3_bias, sizeof(float) * conv4_3_out_channel,
              hipMemcpyHostToDevice);

  //////////BLOCK 5/////////////////////////////////
  hipMemcpy(d_conv5_1_weight, conv5_1_weight,
             sizeof(float) * conv5_1_in_channel * conv5_1_out_channel *
                 conv5_1_kernel_size * conv5_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_1_bias, conv5_1_bias, sizeof(float) * conv5_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_2_weight, conv5_2_weight,
              sizeof(float) * conv5_2_in_channel * conv5_2_out_channel *
                  conv5_2_kernel_size * conv5_2_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_2_bias, conv5_2_bias, sizeof(float) * conv5_2_out_channel,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_3_weight, conv5_3_weight,
              sizeof(float) * conv5_3_in_channel * conv5_3_out_channel *
                  conv5_3_kernel_size * conv5_3_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_3_bias, conv5_3_bias, sizeof(float) * conv5_3_out_channel,
              hipMemcpyHostToDevice);


  hipMemcpy(d_fc1_weight, fc1_weight,
             sizeof(float) * fc1_in_channel * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_bias, fc1_bias, sizeof(float) * fc1_out_channel,
             hipMemcpyHostToDevice);

  // copy input image
  size_t image_size = batch * input_size * input_size * input_channel;
  hipMemcpy(d_image, image, image_size * sizeof(uint8_t),
             hipMemcpyHostToDevice);
}

void vgg16_cuda::classify(int* predict, int batch) {
  // read logits back to cpu
  hipMemcpy(output, d_output, sizeof(float) * output_size * batch,
             hipMemcpyDeviceToHost);
  // Softmax
  softmax(output, predict, batch, output_size);
}

vgg16_cuda::~vgg16_cuda() {
  hipFree(d_conv1_1_weight);   
  hipFree(d_conv1_2_weight);   
  hipFree(d_conv2_1_weight);   
  hipFree(d_conv2_2_weight);  
  hipFree(d_conv3_1_weight);   
  hipFree(d_conv3_2_weight);   
  hipFree(d_conv3_3_weight);   
  hipFree(d_conv4_1_weight);   
  hipFree(d_conv4_2_weight);   
  hipFree(d_conv4_3_weight); 
  hipFree(d_conv5_1_weight);   
  hipFree(d_conv5_2_weight);   
  hipFree(d_conv5_3_weight);   
 
  hipFree(d_conv1_1_bias);   
  hipFree(d_conv1_2_bias);   
  hipFree(d_conv2_1_bias);   
  hipFree(d_conv2_2_bias);  
  hipFree(d_conv3_1_bias);   
  hipFree(d_conv3_2_bias);   
  hipFree(d_conv3_3_bias);   
  hipFree(d_conv4_1_bias);   
  hipFree(d_conv4_2_bias);   
  hipFree(d_conv4_3_bias); 
  hipFree(d_conv5_1_bias);   
  hipFree(d_conv5_2_bias);   
  hipFree(d_conv5_3_bias);   
   
  hipFree(d_fc1_weight);     
  hipFree(d_fc1_bias);        

  hipFree(d_image);          
  hipFree(d_input); 

  hipFree(d_input_padded);          
  hipFree(d_C1_1_feature_map); 
  hipFree(d_C1_1_feature_map_padded); 
  hipFree(d_C1_2_feature_map); 
  hipFree(d_S1_feature_map); 

  hipFree(d_S1_feature_map_padded); 
  hipFree(d_C2_1_feature_map); 
  hipFree(d_C2_1_feature_map_padded); 
  hipFree(d_C2_2_feature_map); 
  hipFree(d_S2_feature_map); 

  hipFree(d_S2_feature_map_padded); 
  hipFree(d_C3_1_feature_map); 
  hipFree(d_C3_1_feature_map_padded); 
  hipFree(d_C3_2_feature_map); 
  hipFree(d_C3_2_feature_map_padded); 
  hipFree(d_C3_3_feature_map); 
  hipFree(d_S3_feature_map); 

  hipFree(d_S3_feature_map_padded); 
  hipFree(d_C4_1_feature_map); 
  hipFree(d_C4_1_feature_map_padded); 
  hipFree(d_C4_2_feature_map); 
  hipFree(d_C4_2_feature_map_padded); 
  hipFree(d_C4_3_feature_map); 
  hipFree(d_S4_feature_map); 

  hipFree(d_S4_feature_map_padded); 
  hipFree(d_C5_1_feature_map); 
  hipFree(d_C5_1_feature_map_padded); 
  hipFree(d_C5_2_feature_map); 
  hipFree(d_C5_2_feature_map_padded); 
  hipFree(d_C5_3_feature_map); 
  hipFree(d_S5_feature_map); 
 
  hipFree(d_output);       
  hipFree(d_predict_cuda);   
}
