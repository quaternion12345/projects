#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <assert.h>
#include "matmul.h"
using namespace std;
__global__ void Kernel(const int* const matrixA, const int* const matrixB, int* const matrixC, const int n);
void allocateDeviceMemory(void** M, int size)
{
  hipError_t err = hipMalloc(M, size);
  assert(err==hipSuccess);
}

void deallocateDeviceMemory(void* M)
{
  hipError_t err = hipFree(M);
  assert(err==hipSuccess);
}

void matmul_ref(const int* const matrixA, const int* const matrixB,
                int* const matrixC, const int n) {
  // You can assume matrixC is initialized with zero
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      for (int k = 0; k < n; k++)
        matrixC[i * n + j] += matrixA[i * n + k] * matrixB[k * n + j];
}

void matmul_optimized(const int* const matrixA, const int* const matrixB,
                      int* const matrixC, const int* const d_A, const int* const d_B,  int* const d_C, const int n) {

  // TODO: Implement your CUDA code
  #define TILE_WIDTH 32
  dim3 dimGrid(ceil((float)n/(float)TILE_WIDTH), ceil((float)n/(float)TILE_WIDTH), 1);
  dim3 dimBlock(TILE_WIDTH/2, TILE_WIDTH/2, 1);

  hipMemcpy((void*)d_A, matrixA, sizeof(int)*n*n, hipMemcpyHostToDevice);
  hipMemcpy((void*)d_B, matrixB, sizeof(int)*n*n, hipMemcpyHostToDevice);
  Kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);
  hipMemcpy(matrixC, d_C, sizeof(int)*n*n, hipMemcpyDeviceToHost);
}

__global__ void Kernel(const int* const matrixA, const int* const matrixB, int* const matrixC, const int n){
  // calculate 4 elements for each thread for speed up, similar to loop unrolling
  // (row,col), (row+TILE_SIZE/2, col), (row, col+TILE_SIZE/2), (row + TILE_SIZE/2, col + TILE_SIZE/2)
  
  // type def
  int width = n;
  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  
  // shared memory
  __shared__ int subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ int subTileB[TILE_WIDTH][TILE_WIDTH];

  // row, col
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  // local sum
  int sum1=0;
  int sum2=0;
  int sum3=0;
  int sum4=0;
  
  // stride
  int diff= TILE_WIDTH / 2;

  // calculate
  for(int l=0; l < ceil((float)width/(float)TILE_WIDTH); ++l){
    // Load shared memory
    // if subTile is out of original matrix, set 0

    // Make SubTileA
    if(row < width && l * TILE_WIDTH + tx < width)
      subTileA[ty][tx] = matrixA[row * width + l * TILE_WIDTH + tx];
    else subTileA[ty][tx] = 0;

    if(row < width && l * TILE_WIDTH + tx + diff < width)
      subTileA[ty][tx + diff] = matrixA[row * width + l * TILE_WIDTH + tx + diff];
    else subTileA[ty][tx + diff] = 0;

    if(row + diff < width && l * TILE_WIDTH + tx < width)
      subTileA[ty + diff][tx] = matrixA[(row + diff) * width + l * TILE_WIDTH + tx];
    else subTileA[ty + diff][tx] = 0;

    if(row + diff < width && l * TILE_WIDTH + tx + diff < width)
      subTileA[ty + diff][tx + diff] = matrixA[(row + diff) * width + l * TILE_WIDTH + tx + diff];
    else subTileA[ty + diff][tx + diff] = 0;

    // Make SubTileB
    if(l * TILE_WIDTH + ty < width && col < width)
      subTileB[ty][tx] = matrixB[(l * TILE_WIDTH + ty) * width + col];
    else subTileB[ty][tx] = 0;

    if(l * TILE_WIDTH + ty < width && col + diff < width)
      subTileB[ty][tx + diff] = matrixB[(l * TILE_WIDTH + ty) * width + col + diff];
    else subTileB[ty][tx + diff] = 0;

    if(l * TILE_WIDTH + ty + diff < width && col < width)
      subTileB[ty + diff][tx] = matrixB[(l * TILE_WIDTH + ty + diff) * width + col];
    else subTileB[ty + diff][tx] = 0;

    if(l * TILE_WIDTH + ty + diff < width && col + diff < width)
      subTileB[ty + diff][tx + diff] = matrixB[(l * TILE_WIDTH + ty + diff) * width + col + diff];
    else subTileB[ty + diff][tx + diff] = 0;

    // synchronize threads
    __syncthreads();

    // Computation
    for(int k=0; k < TILE_WIDTH; ++k){
      sum1 += subTileA[ty][k] * subTileB[k][tx];
      sum2 += subTileA[ty][k] * subTileB[k][tx+diff];
      sum3 += subTileA[ty+diff][k] * subTileB[k][tx];
      sum4 += subTileA[ty+diff][k] * subTileB[k][tx+diff];
    }
    __syncthreads();
  }

  // Write Back
  if(row < width && col < width)
    matrixC[row * width + col] = sum1;
  if(row < width && col + diff < width)
    matrixC[row * width + col + diff] = sum2;
  if(row + diff < width && col < width)
    matrixC[(row + diff) * width + col] = sum3;
  if(row + diff < width && col + diff < width)
    matrixC[(row + diff) * width + col + diff] = sum4;
  
}
